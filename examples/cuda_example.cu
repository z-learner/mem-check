#include <hip/hip_runtime.h>
#include <iostream>

__global__ void simpleKernel(int *d_data) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  d_data[idx] = idx;
}

void testCudaMalloc() {
  int *d_data;
  size_t size = 1024 * sizeof(int);

  hipMalloc(&d_data, size);
  simpleKernel<<<1, 1024>>>(d_data);
  hipDeviceSynchronize();
  hipFree(d_data);
}

void testCudaMallocHost() {
  int *h_data;
  size_t size = 1024 * sizeof(int);

  hipHostMalloc(&h_data, size, hipHostMallocDefault);
  for (int i = 0; i < 1024; ++i) {
    h_data[i] = i;
  }
  hipHostFree(h_data);
}

void testCudaMallocManaged() {
  int *m_data;
  size_t size = 1024 * sizeof(int);

  hipMallocManaged(&m_data, size);
  simpleKernel<<<1, 1024>>>(m_data);
  hipDeviceSynchronize();
  hipFree(m_data);
}

int main() {
  std::cout << "Testing cudaMalloc..." << std::endl;
  testCudaMalloc();

  std::cout << "Testing cudaMallocHost..." << std::endl;
  testCudaMallocHost();

  std::cout << "Testing cudaMallocManaged..." << std::endl;
  testCudaMallocManaged();

  return 0;
}
